#include "cublas_utils.cuh"

std::unique_ptr<hipblasHandle_t, cublas_handle_deleter> cublas_init_handle()
{
  std::unique_ptr<hipblasHandle_t> handle_ptr{new hipblasHandle_t};
  CUBLAS_CHECK(hipblasCreate(handle_ptr.get()));

  return std::unique_ptr<hipblasHandle_t, cublas_handle_deleter>{handle_ptr.release(), cublas_handle_deleter{}};
}

hipblasStatus_t cublasCopy(hipblasHandle_t handle,
                          int n,
                          const float *x,
                          int incx,
                          float *y,
                          int incy)
{
  return hipblasScopy(handle, n, x, incx, y, incy);
}

hipblasStatus_t cublasCopy(hipblasHandle_t handle,
                          int n,
                          const double *x,
                          int incx,
                          double *y,
                          int incy)
{
  return hipblasDcopy(handle, n, x, incx, y, incy);
}

hipblasStatus_t cublasAxpy(hipblasHandle_t handle,
                          int n,
                          const float *alpha, /* host or device pointer */
                          const float *x,
                          int incx,
                          float *y,
                          int incy)
{
  return hipblasSaxpy(handle, n, alpha, x, incx, y, incy);
}

hipblasStatus_t cublasAxpy(hipblasHandle_t handle,
                          int n,
                          const double *alpha, /* host or device pointer */
                          const double *x,
                          int incx,
                          double *y,
                          int incy)
{
  return hipblasDaxpy(handle, n, alpha, x, incx, y, incy);
}

hipblasStatus_t cublasIamax(hipblasHandle_t handle,
                           int n,
                           const float *x,
                           int incx,
                           int *result)

{
  return hipblasIsamax(handle, n, x, incx, result);
}

hipblasStatus_t cublasIamax(hipblasHandle_t handle,
                           int n,
                           const double *x,
                           int incx,
                           int *result)
{
  return hipblasIdamax(handle, n, x, incx, result);
}

hipblasStatus_t cublasGeam(hipblasHandle_t handle,
                          hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n,
                          const float *alpha,
                          const float *A, int lda,
                          const float *beta,
                          const float *B, int ldb,
                          float *C, int ldc)
{
  return hipblasSgeam(handle, transa, transb, m, n, alpha, A, lda, beta, B, ldb, C, ldc);
}

hipblasStatus_t cublasGeam(hipblasHandle_t handle,
                          hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n,
                          const double *alpha,
                          const double *A, int lda,
                          const double *beta,
                          const double *B, int ldb,
                          double *C, int ldc)
{
  return hipblasDgeam(handle, transa, transb, m, n, alpha, A, lda, beta, B, ldb, C, ldc);
}
