#include "cuda_utils.cuh"

std::unique_ptr<hipStream_t[], cu_streams_deleter> cu_create_streams(std::size_t size)
{
  std::unique_ptr<hipStream_t[]> streams_ptr{new hipStream_t[size]};

  for (std::size_t i = 0; i < size; ++i)
  {
    CUDA_CHECK(hipStreamCreate(&streams_ptr[i]));
  }

  return std::unique_ptr<hipStream_t[], cu_streams_deleter>{streams_ptr.release(), cu_streams_deleter{size}};
}

std::unique_ptr<hipStream_t, cu_stream_deleter> cu_create_stream()
{
  std::unique_ptr<hipStream_t> stream_ptr{new hipStream_t};
  CUDA_CHECK(hipStreamCreate(stream_ptr.get()));
  return std::unique_ptr<hipStream_t, cu_stream_deleter>{stream_ptr.release(), cu_stream_deleter{}};
}
