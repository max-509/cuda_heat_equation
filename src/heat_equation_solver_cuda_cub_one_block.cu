#include "hip/hip_runtime.h"
#include "heat_equation_solver_impl.h"

#include "cuda_utils/cuda_utils.cuh"

#include <cub/block/block_reduce.cuh>
#include <cub/device/device_reduce.cuh>

#include <stdexcept>
#include <cstdlib>
#include <cmath>

namespace
{
  constexpr std::size_t GRID_BLOCK_SIZE = 16;
  constexpr std::size_t ERROR_BLOCK_SIZE = 16;
}

#ifndef N_ERR_COMPUTING_IN_DEVICE
#define N_ERR_COMPUTING_IN_DEVICE 1500
#endif // N_ERR_COMPUTING_IN_DEVICE

template <int block_size>
__global__ void compute_partial_errors(const FLOAT_TYPE *__restrict__ curr_grid,
                                 const FLOAT_TYPE *__restrict__ next_grid,
                                 const std::size_t grid_size,
                                 const std::size_t pitch,
                                 FLOAT_TYPE *__restrict__ err_ptr)
{
  using BlockReduce = hipcub::BlockReduce<FLOAT_TYPE, block_size, hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY, block_size>;

  __shared__ typename BlockReduce::TempStorage temp_storage;
  const auto thread_data_size = grid_size / block_size + ((grid_size % block_size) != 0);
  const auto i_y_begin = threadIdx.y * thread_data_size;
  const auto i_y_end = min(grid_size, i_y_begin + thread_data_size);
  const auto i_x_begin = threadIdx.x * thread_data_size;
  const auto i_x_end = min(grid_size, i_x_begin + thread_data_size);

  FLOAT_TYPE abs_diff = 0.0;

  #pragma unroll
  for (std::size_t i_y = i_y_begin; i_y < i_y_end; ++i_y)
  {
    for (std::size_t i_x = i_x_begin; i_x < i_x_end; ++i_x)
    {
      const auto grid_idx = i_y * pitch + i_x;
      abs_diff = fmax(abs_diff, fabs(curr_grid[grid_idx] - next_grid[grid_idx]));
    }
  }

  auto block_max_err = BlockReduce(temp_storage).Reduce(abs_diff, hipcub::Max{});

  if (threadIdx.x == 0)
  {
    *err_ptr = block_max_err;
  }
}

__global__ void grid_recompute(const FLOAT_TYPE *__restrict__ curr_grid,
                               FLOAT_TYPE *__restrict__ next_grid,
                               const std::size_t grid_size,
                               const std::size_t pitch)
{
  unsigned int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y_idx = blockIdx.y * blockDim.y + threadIdx.y;

  if ((x_idx > 0 && x_idx < grid_size - 1) &&
      (y_idx > 0 && y_idx < grid_size - 1))
  {
    std::size_t grid_idx = y_idx * pitch + x_idx;
    next_grid[grid_idx] = (FLOAT_TYPE)0.25 * (curr_grid[grid_idx - pitch] +
                                              curr_grid[grid_idx + pitch] +
                                              curr_grid[grid_idx - 1] +
                                              curr_grid[grid_idx + 1]);
  }
}

int solve_heat_equation(FLOAT_TYPE *__restrict__ init_grid,
                        const size_t grid_size,
                        const size_t max_iter,
                        const FLOAT_TYPE etol,
                        size_t *last_iter,
                        FLOAT_TYPE *last_etol)
{
  const std::size_t grid_sqr = grid_size * grid_size;
  auto cu_streams_unique = cu_create_streams(2);

  std::size_t grid_pitch;
  auto cu_buff_grid_unique = cu_make_pitched_memory_unique<FLOAT_TYPE>(2 * grid_size, grid_size, grid_pitch);

  auto cu_init_grid_pinned_unique = cu_make_pinned_memory_unique(init_grid, grid_sqr);

  const std::size_t half_buff_grid_size = grid_size * grid_pitch;
  CUDA_CHECK(cu_memcpy2D_async(cu_buff_grid_unique.get(), grid_pitch,
                               cu_init_grid_pinned_unique.get(), grid_size,
                               grid_size, grid_size,
                               hipMemcpyDefault, cu_streams_unique[0]));
  CUDA_CHECK(cu_memcpy2D_async(cu_buff_grid_unique.get() + half_buff_grid_size, grid_pitch,
                               cu_init_grid_pinned_unique.get(), grid_size,
                               grid_size, grid_size,
                               hipMemcpyDefault, cu_streams_unique[1]));

  CUDA_CHECK(hipDeviceSynchronize());

  auto cu_err_unique = cu_make_host_memory_unique<FLOAT_TYPE>();
  FLOAT_TYPE &err = *cu_err_unique.get();
  err = INFINITY;

  size_t curr_iter;
  size_t n_err_iter;
  dim3 kernel_threads(GRID_BLOCK_SIZE, GRID_BLOCK_SIZE);
  dim3 kernel_blocks((grid_size / kernel_threads.x) + (grid_size % kernel_threads.x != 0),
                     (grid_size / kernel_threads.y) + (grid_size % kernel_threads.y != 0));
  for (curr_iter = 0u; curr_iter < max_iter && err > etol; curr_iter += n_err_iter)
  {

    for (n_err_iter = 0; n_err_iter < N_ERR_COMPUTING_IN_DEVICE; n_err_iter += 2)
    {

      grid_recompute<<<kernel_blocks, kernel_threads, 0, cu_streams_unique[0]>>>(
          cu_buff_grid_unique.get(), cu_buff_grid_unique.get() + half_buff_grid_size, grid_size, grid_pitch);

      grid_recompute<<<kernel_blocks, kernel_threads, 0, cu_streams_unique[0]>>>(
          cu_buff_grid_unique.get() + half_buff_grid_size, cu_buff_grid_unique.get(), grid_size, grid_pitch);
    }

    compute_partial_errors<ERROR_BLOCK_SIZE><<<1, dim3(ERROR_BLOCK_SIZE, ERROR_BLOCK_SIZE), 0, cu_streams_unique[0]>>>(
        cu_buff_grid_unique.get(), cu_buff_grid_unique.get() + half_buff_grid_size, grid_size, grid_pitch, &err);

    CUDA_CHECK(hipStreamSynchronize(cu_streams_unique[0]));
  }

  CUDA_CHECK(cu_memcpy2D_async(cu_init_grid_pinned_unique.get(), grid_size,
                               cu_buff_grid_unique.get(), grid_pitch,
                               grid_size, grid_size, hipMemcpyDefault));

  if (NULL != last_iter)
  {
    *last_iter = curr_iter;
  }
  if (NULL != last_etol)
  {
    *last_etol = err;
  }

  return 0;
}

const char *get_solver_version()
{
  return "CUDA CUB one block";
}
